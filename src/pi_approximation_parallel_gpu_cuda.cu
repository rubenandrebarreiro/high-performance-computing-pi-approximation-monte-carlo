#include "hip/hip_runtime.h"
// Aproximacao de PI (com animacao) - Versao Paralela em GPU com CUDA
// Demonstracao para Arquitetura de Sistemas de Computadores (ASC)
// Ciencias - ULisboa (2024/2025 - Semestre 1)
// @author: Ruben Andre Barreiro

// Importacao das bibliotecas necessarias

// Importacao do modulo de Graficos da biblioteca
// Simple and Fast Multimedia Library (SFML)
// ReSharper disable All
#include <SFML/Graphics.hpp>

// Importacao da biblioteca I/O Stream
// para escrita e leitura de dados
#include <iostream>

// Importacao da biblioteca String Stream
// para escrita de cadeias de caracteres (strings)
#include <sstream>

// Importacao da biblioteca principal
// do CUDA para programacao em GPU
#include <hip/hip_runtime.h>

// Importacao da biblioteca de Tempo de Execucao
// do CUDA para programacao em GPU
#include <hip/hip_runtime.h>

// Importacao da biblioteca da API
// de Tempo de Execucao do CUDA para programacao em GPU
#include <hip/hip_runtime_api.h>

// Importacao da biblioteca CURAND (CUDA RANDOM)
// para a geracao de numeros pseudo-aleatorios em GPU
#include <hiprand/hiprand_kernel.h>


// Definicao/Re-Definicao das diretrizes em CUDA
// a serem usadas ao longo da execucao do programa
#ifndef __HIPCC__
    #define __device__
    #define __host__
    #define __global__
    #define __syncthreads()
    #define blockIdx (dim3{0,0,0})
    #define threadIdx (dim3{0,0,0})
    #define blockDim (dim3{0,0,0})
    #define gridDim (dim3{0,0,0})
#endif


// Definicao do kernel em CUDA para gerar os pontos pseudo-aleatoriamente e
// contar o numero de pontos pseudo-aleatorios que estao no interior da circunferencia
__global__ void generate_random_points_and_count_inside_circle(int* inside_circle_count,
                                                               double* x_points, double* y_points,
                                                               const bool is_partial,
                                                               const int CUDA_BLOCK_SIZE,
                                                               const int NUM_POINTS, const double RADIUS) {

    // Definicao de um bloco de dados partilhado no dispositivo (GPU)
    extern volatile __shared__ unsigned int cuda_block_shared_data[];


    // Definicao do indice do thread local lancado pelo dispositivo (GPU)
    unsigned int local_cuda_thread_idx =
        threadIdx.x;

    // Definicao do indice do bloco de dados lancado pelo dispositivo (GPU)
    unsigned int cuda_block_idx =
        blockIdx.x;

    // Definicao da dimensao da grelha de dados lancada pelo dispositivo (GPU)
    unsigned int cuda_grid_dim =
        gridDim.x;

    // Definicao do indice do thread global lancado pelo dispositivo (GPU)
    unsigned int global_cuda_thread_idx =
            ( ( cuda_block_idx * ( 2 * CUDA_BLOCK_SIZE ) ) +
                local_cuda_thread_idx );

    // Definicao do tamanho da grelha de dados lancada pelo dispositivo (GPU)
    unsigned int cuda_grid_size =
        ( 2 * CUDA_BLOCK_SIZE * cuda_grid_dim );


    // Inicializacao do bloco de dados partilhado no dispositivo (GPU)
    cuda_block_shared_data[local_cuda_thread_idx] = 0;


    // Se o indice do thread global lancado pelo dispositivo (GPU)
    // e menor que o numero total de pontos a serem gerados de forma
    // pseudo-aleatoria e se esta a ser tratada uma reducao parcial
    if(global_cuda_thread_idx < NUM_POINTS && is_partial) {

        // Definicao do estado de um gerador pseudo-aleatorio do CURAND
        hiprandState prng_state;

        // Definicao do gerador pseudo-aleatorio do CURAND,
        // para a geracao das coordenadas x e y dos pontos
        hiprand_init(clock(), global_cuda_thread_idx,
                    0, &prng_state);


        // Geracao pseudo-aleatoria da coordenadas x no intervalo [-RADIUS, RADIUS]
        const double x = -RADIUS + hiprand_uniform(&prng_state) * 2 * RADIUS;

        // Geracao pseudo-aleatoria da coordenadas y no intervalo [-RADIUS, RADIUS]
        const double y = -RADIUS + hiprand_uniform(&prng_state) * 2 * RADIUS;


        // Copia da coordenada pseudo-aleatoria x
        // do ponto (x,y) para a memoria do dispositivo (GPU)
        x_points[global_cuda_thread_idx] = x;

        // Copia da coordenada pseudo-aleatoria y
        // do ponto (x,y) para a memoria do dispositivo (GPU)
        y_points[global_cuda_thread_idx] = y;


        // Verificacao sobre se o ponto (x,y) gerado pseudo-aleatoriamente
        // esta no interior da circunferencia de raio RADIUS
        if (x * x + y * y <= RADIUS * RADIUS) {

            // Contagem do ponto interior a circunferencia de raio RADIUS
            inside_circle_count[global_cuda_thread_idx] += 1;

        }

    }

    // Sincronizacao dos CUDA threads lancados no dispositivo (GPU)
    __syncthreads();


    // Enquanto o indice do thread global lancado pelo dispositivo (GPU)
    // e menor que o numero maximo de pontos a serem gerados de forma pseudo-aleatoria
    while(global_cuda_thread_idx < NUM_POINTS) {

        // Calculo do indice do thread global lancado pelo dispositivo (GPU),
        // considerando um offset equivalente ao tamanho do bloco de dados
        // a ser considerado pelo dispositivo (GPU)
        unsigned int global_cuda_thread_idx_shifted_with_cuda_block_offset =
                ( global_cuda_thread_idx + CUDA_BLOCK_SIZE );

        // Reducao (soma) da contagem dos pontos interiores em relacao
        // ao indice do thread global lancado pelo dispositivo (GPU),
        // e ao mesmo em relacao a vizinhanca com offset do tamanho
        // do bloco de dados a ser considerado pelo dispositivo (GPU),
        // atualizando o bloco de dados partilhado no dispositivo (GPU)
        cuda_block_shared_data[local_cuda_thread_idx] +=
            inside_circle_count[global_cuda_thread_idx] +
                inside_circle_count[global_cuda_thread_idx_shifted_with_cuda_block_offset];

        // Soma do tamanho da grelha de dados a ser lancada pelo dispositivo (GPU)
        // ao indice do thread global lancado pelo dispositivo (GPU)
        global_cuda_thread_idx += cuda_grid_size;

    }

    // Sincronizacao dos CUDA threads lancados no dispositivo (GPU)
    __syncthreads();


    // Se o tamanho do bloco de dados a ser considerado
    // pelo dispositivo (GPU) e maior ou igual a 512
    if(CUDA_BLOCK_SIZE >= 512) {

        // Se o indice do thread local lancado
        // pelo dispositivo (GPU) e menor que 256
        if (local_cuda_thread_idx < 256) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 256
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 256];

        }

        // Sincronizacao dos CUDA threads lancados no dispositivo (GPU)
        __syncthreads();

    }

    // Se o tamanho do bloco de dados a ser considerado
    // pelo dispositivo (GPU) e maior ou igual a 256
    if(CUDA_BLOCK_SIZE >= 256) {

        // Se o indice do thread local lancado
        // pelo dispositivo (GPU) e menor que 128
        if (local_cuda_thread_idx < 128) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 128
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 128];

        }

        // Sincronizacao dos CUDA threads lancados no dispositivo (GPU)
        __syncthreads();

    }

    // Se o tamanho do bloco de dados a ser considerado
    // pelo dispositivo (GPU) e maior ou igual a 128
    if(CUDA_BLOCK_SIZE >= 128) {

        // Se o indice do thread local lancado
        // pelo dispositivo (GPU) e menor que 64
        if (local_cuda_thread_idx < 64) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 64
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 64];

        }

        // Sincronizacao dos CUDA threads lancados no dispositivo (GPU)
        __syncthreads();

    }


    // Se o indice do thread local lancado
    // pelo dispositivo (GPU) e menor que 32
    if (local_cuda_thread_idx < 32) {

        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 64
        if(CUDA_BLOCK_SIZE >= 64) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 32
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 32];

        }


        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 32
        if (CUDA_BLOCK_SIZE >= 32) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 16
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 16];

        }


        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 16
        if (CUDA_BLOCK_SIZE >= 16) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 8
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 8];

        }


        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 8
        if (CUDA_BLOCK_SIZE >= 8) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 4
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 4];

        }


        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 4
        if (CUDA_BLOCK_SIZE >= 4) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 2
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 2];

        }


        // Se o tamanho do bloco de dados a ser considerado
        // pelo dispositivo (GPU) e maior ou igual a 2
        if (CUDA_BLOCK_SIZE >= 2) {

            // Reducao (soma) da contagem do bloco de dados partilhado
            // no dispositivo (GPU) em relacao a vizinhanca com offset 1
            cuda_block_shared_data[local_cuda_thread_idx] +=
                cuda_block_shared_data[local_cuda_thread_idx + 1];

        }

    }


    // Se o indice do thread local lancado pelo dispositivo (GPU)
    // corresponde ao primeiro thread local lancado
    if (local_cuda_thread_idx == 0) {

        // Reducao da contagem de um ponto interior a circunferencia
        // em relacao ao bloco de dados partilhado no dispositivo (GPU)
        inside_circle_count[blockIdx.x] =
            cuda_block_shared_data[0];

    }

}


// Definicao da funcao principal (main) do programa
int main() {

    // Definicao de algumas configuracoes
    // iniciais para o programa

    // Definicao da constante para guardar
    // o numero maximo de pontos que podem ser gerados
    constexpr int NUM_MAX_POINTS = 400000;

    // Definicao da constante para guardar
    // o valor real de PI
    constexpr double PI_REAL = 3.14159;

    // Definicao da constante para guardar
    // o raio da circunferencia
    constexpr float RADIUS = 2.0;

    // Definicao do tamanho do bloco de dados
    // a ser considerado pelo dispositivo (GPU)
    constexpr int CUDA_BLOCK_SIZE = 256;

    // Definicao do tamanho da grelha de dados
    // a ser considerada pelo dispositivo (GPU)
    const int CUDA_GRID_SIZE =
        ceil( static_cast<double>(NUM_MAX_POINTS) / CUDA_BLOCK_SIZE );


    // Alocacao de memoria no host (CPU) para as coordenadas x
    // dos pontos a serem gerados de forma pseudo-aleatoria
    auto* x_points_host_cpu = new double[NUM_MAX_POINTS];

    // Alocacao de memoria no host (CPU) para as coordenadas y
    // dos pontos a serem gerados de forma pseudo-aleatoria
    auto* y_points_host_cpu = new double[NUM_MAX_POINTS];

    // Definicao do numero de pontos interiores a circunferencia
    // a serem contabilizados na memoria do host (CPU)
    auto num_points_inside_circle_host_cpu =
        static_cast<int *>( malloc( sizeof(int) ) );


    // Definicao das coordenadas x dos pontos a serem gerados
    // de forma pseudo-aleatoria na memoria do dispositivo (GPU)
    double* x_points_device_gpu;

    // Definicao das coordenadas y dos pontos a serem gerados
    // de forma pseudo-aleatoria na memoria do dispositivo (GPU)
    double* y_points_device_gpu;

    // Definicao do numero de pontos interiores a circunferencia
    // a serem contabilizados na memoria do dispositivo (GPU)
    int* num_points_inside_circle_device_gpu;


    // Alocacao de memoria no dispositivo (GPU) para as coordenadas x
    // dos pontos a serem gerados de forma pseudo-aleatoria
    hipMalloc(reinterpret_cast<void **>(&x_points_device_gpu),
               NUM_MAX_POINTS * sizeof(double));

    // Alocacao de memoria no dispositivo (GPU) para as coordenadas y
    // dos pontos a serem gerados de forma pseudo-aleatoria
    hipMalloc(reinterpret_cast<void **>(&y_points_device_gpu),
               NUM_MAX_POINTS * sizeof(double));

    // Alocacao de memoria no dispositivo (GPU) para o numero
    // de pontos interiores a circunferencia a serem contabilizados
    hipMalloc(reinterpret_cast<void **>(&num_points_inside_circle_device_gpu),
               NUM_MAX_POINTS * sizeof(int));


    // Inicializacao do valor a 0 para o numero de pontos
    // interiores a circunferencia a serem contabilizados
    // na memoria do dispositivo (GPU)
    hipMemset(num_points_inside_circle_device_gpu,
               0, NUM_MAX_POINTS * sizeof(int));


    // Chamada ao kernel em CUDA para gerar os pontos pseudo-aleatoriamente e
    // contar o numero de pontos pseudo-aleatorios que estao no interior
    // da circunferencia, considerando um padrao de reducao parcial
    // NOTA: Os numeros pseudo-aleatorios so sao gerados nas reducoes parciais
    generate_random_points_and_count_inside_circle
            <<<CUDA_GRID_SIZE, CUDA_BLOCK_SIZE, ( CUDA_BLOCK_SIZE * sizeof( int ) ) >>>
              (num_points_inside_circle_device_gpu,
               x_points_device_gpu, y_points_device_gpu,
               true, CUDA_BLOCK_SIZE,
               NUM_MAX_POINTS, RADIUS);


    // Sincronizacao dos CUDA threads lancados pelo dispositivo (GPU)
    hipDeviceSynchronize();


    // Chamada ao kernel em CUDA para gerar os pontos pseudo-aleatoriamente e
    // contar o numero de pontos pseudo-aleatorios que estao no interior
    // da circunferencia, considerando um padrao de reducao final
    // (considerando os numeros pseudo-aleatorios gerados anteriormente)
    // NOTA: Os numeros pseudo-aleatorios so sao gerados nas reducoes parciais
    generate_random_points_and_count_inside_circle
            <<<1, CUDA_BLOCK_SIZE, ( CUDA_BLOCK_SIZE * sizeof( int ) ) >>>
              (num_points_inside_circle_device_gpu,
               x_points_device_gpu, y_points_device_gpu,
               false, CUDA_BLOCK_SIZE,
               NUM_MAX_POINTS, RADIUS);


    // Copia as coordenadas x dos pontos gerados anteriormente
    // de forma pseudo-aleatoria na memoria do dispositivo (GPU)
    // para a memoria do host (CPU)
    hipMemcpy(x_points_host_cpu, x_points_device_gpu,
               NUM_MAX_POINTS * sizeof(double),
               hipMemcpyDeviceToHost);

    // Copia as coordenadas y dos pontos gerados anteriormente
    // de forma pseudo-aleatoria na memoria do dispositivo (GPU)
    // para a memoria do host (CPU)
    hipMemcpy(y_points_host_cpu, y_points_device_gpu,
               NUM_MAX_POINTS * sizeof(double),
               hipMemcpyDeviceToHost);

    // Copia o numero de pontos interiores a circunferencia
    // contabilizados na memoria do dispositivo (GPU)
    // para a memoria do host (CPU)
    hipMemcpy(&num_points_inside_circle_host_cpu[0],
               &num_points_inside_circle_device_gpu[0],
               sizeof(int), hipMemcpyDeviceToHost);


    // Inicialização do vetor de pontos interiores a circunferencia
    std::vector<sf::CircleShape> inside_points;

    // Inicialização do vetor de pontos exteriores a circunferencia
    std::vector<sf::CircleShape> outside_points;


    // Criar a janela de renderizacao para a aproximacao de PI,
    // usando a biblioteca Simple and Fast Multimedia Library (SFML)
    sf::RenderWindow window(sf::VideoMode(800, 600),
                            "Aproximacao de PI - Animacao "
                            "(Versao Paralela em GPU com CUDA)");

    // Desenho da circunferencia com o raio associado
    // (representando a area da aproximacao de PI),
    // usando um factor de 100 para ajustar
    // o tamanho da circunferencia
    sf::CircleShape circle(RADIUS * 100);

    // Centralizar a circunferencia na disposicao da janela
    circle.setPosition(400 - RADIUS * 100, 300 - RADIUS * 100);

    // Definicao da cor interior como transparente
    circle.setFillColor(sf::Color::Transparent);

    // Definicao da cor da linha exterior (contorno)
    // da circunferencia como preto
    circle.setOutlineColor(sf::Color::Black);

    // Definicao da expessura da linha exterior (contorno)
    // da circunferencia com um factor de 2
    circle.setOutlineThickness(2);


    // Criacao da fonte (tipo de letra) para o texto
    // a definir para a exibicao da estimativa de PI
    sf::Font font;

    // Se nao e possivel carregar a fonte (tipo de letra) para o texto
    // a definir para a exibicao da estimativa de PI, como Arial
    if(!font.loadFromFile("fonts/arial.ttf")) {

        // Criacao de uma cadeia de caracteres (string)
        // para o erro respetivo ao carregamento
        // da fonte (tipo de letra)
        std::cerr << "Falha ao carregar a fonte!" << std::endl;

        // Retorno do valor de flag (etiqueta) a -1 (erro)
        return -1;

    }


    // Criacao de um texto para a informacao
    // sobre a aproximacao de PI
    sf::Text pi_approx_text;

    // Definicao da fonte (tipo de letra) para
    // o texto da informacao sobre a aproximacao de PI
    pi_approx_text.setFont(font);

    // Definicao do tamanho do caracter do texto
    // para a informacao sobre a aproximacao de PI
    pi_approx_text.setCharacterSize(24);

    // Definicao da cor do texto (preto)
    // para a informacao sobre a aproximacao de PI
    pi_approx_text.setFillColor(sf::Color::Black);

    // Definicao da posicao na janela do texto
    // para a informacao sobre a aproximacao de PI
    pi_approx_text.setPosition(10, 520);


    // Criacao de um texto para a informacao
    // sobre o valor real de PI
    sf::Text pi_real_text;

    // Definicao da fonte (tipo de letra) para
    // o texto da informacao sobre o valor real de PI
    pi_real_text.setFont(font);

    // Definicao do tamanho do caracter do texto
    // para a informacao sobre o valor real de PI
    pi_real_text.setCharacterSize(24);

    // Definicao da cor do texto (preto)
    // para a informacao sobre o valor real de PI
    pi_real_text.setFillColor(sf::Color::Black);

    // Definicao da posicao na janela do texto
    // para a informacao sobre o valor real de PI
    pi_real_text.setPosition(10, 550);


    // Definicao do indice correspondente ao ponto atual gerado
    unsigned int current_point = 0;

    // Definicao do valor da aproximacao de PI atual gerada
    double pi_approx = 0.0;


    // Enquanto a janela estiver aberta e nao tenham sido
    // gerados o numero maximo de pontos que podem ser gerados e
    // a aproximacao de PI for diferente do valor real de PI
    while (window.isOpen() && current_point < NUM_MAX_POINTS &&
           pi_approx != PI_REAL) {

        // Para cada ponto a serem gerado por cada batch (lote)
        // associado aos threads lancados pelo dispositivo (GPU)
        for (int i = 0; i < NUM_MAX_POINTS; i++) {

            // Definicao do ponto base a desenhar na tela
            sf::CircleShape point(1);

            // Definicao (e escala) do ponto base a desenhar na tela
            point.setPosition(static_cast<float>(400 + x_points_host_cpu[i] * 100),
                              static_cast<float>(300 + y_points_host_cpu[i] * 100));


            // Se o ponto atual gerado esta dentro da circunferencia
            // (ou seja, o ponto atual gerado e um ponto interno)
            if (x_points_host_cpu[i] * x_points_host_cpu[i] +
                y_points_host_cpu[i] * y_points_host_cpu[i] <= RADIUS * RADIUS) {

                // Definicao da cor do ponto interno como verde
                point.setFillColor(sf::Color::Green);

                // Adicao do ponto atual gerado ao vetor
                // de pontos interiores a circunferencia
                inside_points.push_back(point);

            }
            else {

                // Definicao da cor do ponto externo como vermelho
                point.setFillColor(sf::Color::Red);

                // Adicao do ponto atual gerado ao vetor
                // de pontos exteriores a circunferencia
                outside_points.push_back(point);

            }

        }

        // Soma do indice correspondente ao ponto atual gerado,
        // tendo em consideracao o numero de pontos gerados local
        current_point += NUM_MAX_POINTS;

        // Atualizacao da aproximacao de PI atual,
        // considerando o ponto atual gerado
        pi_approx = 4.0 * num_points_inside_circle_host_cpu[0] / current_point;



        // Criacao de um evento no contexto da biblioteca
        // Simple and Fast Multimedia Library (SFML)
        sf::Event event{};


        // Enquanto a janela espera por um evento
        // (definicao de um "escutador" de eventos)
        while (window.pollEvent(event)) {

            // Se o evento "escutado" for do tipo "fecho"
            if (event.type == sf::Event::Closed) {

                // Fecho da janela criada anteriormente
                window.close();

            }

        }


        // Limpeza da janela e definicao da cor
        // para a janela como branco
        window.clear(sf::Color::White);


        // Definicao do Stream de Cadeia de Caracteres (String)
        // para a aproximacao de PI
        std::stringstream pi_approx_string_stream;

        // Definicao do Stream de Cadeia de Caracteres (String)
        // para o valor real de PI
        std::stringstream pi_real_string_stream;

        // Definicao de escrita do texto da aproximacao de PI,
        // usando o canal de I/O definido anterior
        pi_approx_string_stream << "Estimativa/Aproximacao de PI: "
                                << pi_approx;


        // Definicao de escrita do texto do valor real de PI,
        // usando o canal de I/O definido anterior
        pi_real_string_stream << "Valor Real de PI: "
                              << PI_REAL;


        // Definicao da cadeia de caracteres (String)
        // com o texto da aproximacao de PI, atraves
        // do canal de I/O definido anterior
        pi_approx_text.setString(pi_approx_string_stream.str());

        // Definicao da cadeia de caracteres (String)
        // com o texto do valor real de PI, atraves
        // do canal de I/O definido anterior
        pi_real_text.setString(pi_real_string_stream.str());


        // Para todos os pontos internos a circunferencia
        for (const auto& inside_point : inside_points) {

            // Desenho do ponto atual interno a circunferencia
            window.draw(inside_point);

        }


        // Para todos os pontos externos a circunferencia
        for (const auto& outside_point : outside_points) {

            // Desenho do ponto atual externo a circunferencia
            window.draw(outside_point);

        }


        // Desenho da circunferencia
        window.draw(circle);


        // Desenho do texto para a aproximacao de PI
        window.draw(pi_approx_text);

        // Desenho do texto para o valor real de PI
        window.draw(pi_real_text);


        // Exibicao de todos os graficos definidos
        // anteriormente na janela correspondente
        window.display();


        // Espera de um pequeno tempo antes de adicionar
        // o ponto atual gerado (para efeitos de animacao)
        // NOTA: E possivel ajustar o valor para controlar
        //       a velocidade da animacao (valor por defeito: 1 ms)
        sleep(sf::milliseconds(1));

    }


    // Remove a memoria alocada no host (CPU) para as coordenadas x
    // dos pontos a serem gerados de forma pseudo-aleatoria
    delete[] x_points_host_cpu;

    // Remove a memoria alocada no host (CPU) para as coordenadas y
    // dos pontos a serem gerados de forma pseudo-aleatoria
    delete[] y_points_host_cpu;


    // Remove a memoria alocada no dispositivo (GPU) para as coordenadas x
    // dos pontos a serem gerados de forma pseudo-aleatoria
    hipFree(x_points_device_gpu);

    // Remove a memoria alocada no dispositivo (GPU) para as coordenadas y
    // dos pontos a serem gerados de forma pseudo-aleatoria
    hipFree(y_points_device_gpu);


    // Remove a memoria alocada no dispositivo (GPU) para o numero de
    // pontos interiores a circunferencia a serem contabilizados
    hipFree(num_points_inside_circle_device_gpu);


    // Impressao da estimativa/aproximacao de PI
    printf("Estimativa/Aproximacao de PI: %f\n", pi_approx);

    // Impressao do valor real de PI
    printf("Valor Real de PI: %f\n", PI_REAL);


    // Retorno do valor de flag (etiqueta) a 1 (sucesso)
    return 0;

}
